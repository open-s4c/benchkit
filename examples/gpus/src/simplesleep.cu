#include <iostream>
#include <chrono>
#include <thread>
#include <hip/hip_runtime.h>

// Simple kernel: increments each element
__global__ void simpleKernel(int *data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] += 1;
    }
}

// Simple kernel with artificial delay
__global__ void delayedKernel1(int *data, int size, int delay_iters = 1000000) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int tmp = data[idx];
        // Artificial work: spin in a loop
        for (int i = 0; i < delay_iters; ++i) {
            tmp += i % 7; // just some computation
        }
        data[idx] = tmp;
    }
}

// Simple kernel with artificial delay
__global__ void delayedKernel2(int *data, int size, int delay_iters = 1000000 / 2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int tmp = data[idx];
        // Artificial work: spin in a loop
        for (int i = 0; i < delay_iters; ++i) {
            tmp += i % 7; // just some computation
        }
        data[idx] = tmp;
    }
}

// Helper to print timestamp since start
auto get_relative_time(std::chrono::steady_clock::time_point start) {
    auto now = std::chrono::steady_clock::now();
    auto us = std::chrono::duration_cast<std::chrono::microseconds>(now - start).count();
    return us / 1000.0; // convert to milliseconds
}

int main() {
    const int size = 1 << 20; // 1M elements
    const int bytes = size * sizeof(int);

    int *h_data = new int[size];
    for (int i = 0; i < size; i++) {
        h_data[i] = i;
    }

    int *d_data;
    hipMalloc(&d_data, bytes);
    hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    // Start the timeline
    auto start_time = std::chrono::steady_clock::now();

    std::cout << "[" << get_relative_time(start_time) << " ms] Launching kernel 1..." << std::endl;
    delayedKernel1<<<blocks, threads>>>(d_data, size);
    hipDeviceSynchronize();

    std::cout << "[" << get_relative_time(start_time) << " ms] Sleeping..." << std::endl;
    std::this_thread::sleep_for(std::chrono::milliseconds(500));

    std::cout << "[" << get_relative_time(start_time) << " ms] Launching kernel 2..." << std::endl;
    delayedKernel2<<<blocks, threads>>>(d_data, size);
    hipDeviceSynchronize();

    hipMemcpy(h_data, d_data, bytes, hipMemcpyDeviceToHost);

    std::cout << "[" << get_relative_time(start_time) << " ms] Checking results..." << std::endl;
    std::cout << "Sample results: ";
    for (int i = 0; i < 5; i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << "..." << std::endl;

    hipFree(d_data);
    delete[] h_data;

    std::cout << "[" << get_relative_time(start_time) << " ms] Done!" << std::endl;

    return 0;
}
