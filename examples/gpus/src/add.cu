/*
 * This file is based on the tutorial available on NVIDIA website:
 * https://developer.nvidia.com/blog/even-easier-introduction-cuda/
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif /* BLOCK_SIZE */

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<25;
  float *x, *y;
  const int blockSize = BLOCK_SIZE;
  const int numBlocks = (N + blockSize - 1) / blockSize;

  std::cout << "Add vector benchmark." << std::endl;
  std::cout << "Inputs:" << std::endl;
  std::cout << "  blockSize: " << blockSize << std::endl;
  std::cout << "  numBlocks: " << numBlocks << std::endl;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  auto start = std::chrono::high_resolution_clock::now();

  // Run kernel on N elements on the GPU
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  auto stop = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed_time = stop - start;

  // Check for errors (all values should be 3.0f)
  std::cout << "Outputs:" << std::endl;
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "  max_error: " << maxError << std::endl;
  std::cout << "  kernel_compute_seconds: " << elapsed_time.count() << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
