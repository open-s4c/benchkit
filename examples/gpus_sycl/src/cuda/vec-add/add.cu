// https://developer.nvidia.com/blog/even-easier-introduction-cuda/


#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <chrono>
#include <math.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024 // max
#endif

__global__
void add(int n, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
        y[i] = x[i] + y[i];
}

int main(void) {
    int n = 1<<25;
    float *x, *y;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));

    for (int i=0; i < n; i++) {
        x[i] = 1.0f;
        y[i] = 1.0f;
    }

    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    printf("%d %d\n", BLOCK_SIZE, numBlocks);
    // test events
    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);
    
    auto start = std::chrono::high_resolution_clock::now();
    hipEventRecord(e_start, 0);
    add<<<numBlocks, BLOCK_SIZE>>>(n, x, y);
    hipEventRecord(e_stop, 0);
    hipEventSynchronize(e_stop);
    // cudaDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();
    hipError_t code = hipPeekAtLastError();
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
    }

    std::chrono::duration<double> duration = stop - start;
    std::cout << "duration: " << duration.count()*1000.0f << std::endl;
    float kernel_time = 0;
    hipEventElapsedTime(&kernel_time, e_start, e_stop);
    std::cout << "kernel_time: " << kernel_time << std::endl;

    hipFree(x);
    hipFree(y);
}