

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <assert.h>
#include <math.h>
#include <fcntl.h>
#include <unistd.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32 // max 1024 threads per block
#endif

__global__
void matmul(int n, int *a, int *b, int *c) {
    __shared__ int a_work[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ int b_work[BLOCK_SIZE*BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    // global idx
    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    int tmp = 0;
    // i -> 1024/16 = 64 blocks across
    for (int i=0; i<n/BLOCK_SIZE; i++) {
        a_work[ty*BLOCK_SIZE+tx] = a[y*n + i*BLOCK_SIZE + tx];
        b_work[ty*BLOCK_SIZE+tx] = b[(ty+i*BLOCK_SIZE)*n + x];

        __syncthreads(); // wait for full block

        for (int j=0; j<BLOCK_SIZE; j++) {
            tmp += a_work[ty*BLOCK_SIZE+j] * b_work[j*BLOCK_SIZE+tx];
        }
        __syncthreads();
    }
    c[y*n+x] = tmp;
}

void verify(int n, int *a, int *b, int *c) {
    for (int y=0; y<n; y++) {
        for (int x=0; x<n; x++) {
            int tmp = 0;
            for (int i=0; i<n; i++) {
                tmp += a[n*y+i] * b[n*i+x];
            }
            assert(tmp == c[y*n+x]);
        }
    }
}

int main(void) {
    int n = 1024;
    int size = n*n;

    // host memory
    int *h_a = (int*)malloc(size*sizeof(int));
    int *h_b = (int*)malloc(size*sizeof(int));
    int *h_c = (int*)calloc(size, sizeof(int));

    // device memory
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size*sizeof(int));
    hipMalloc(&d_b, size*sizeof(int));
    hipMalloc(&d_c, size*sizeof(int));

    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(numBlocks, numBlocks, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    printf("%d %d\n", BLOCK_SIZE, numBlocks);

    for (int i=0; i<size; i++) {
        h_a[i] = 1;//i;
        h_b[i] = 1;//i;
    }
    // copy over
    hipMemcpy(d_a, h_a, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size*sizeof(int), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    matmul<<<dimGrid, dimBlock>>>(n, d_a, d_b, d_c);
    hipDeviceSynchronize();
    auto stop = std::chrono::high_resolution_clock::now();

    hipError_t code = hipPeekAtLastError();
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
    }

    std::chrono::duration<double> duration = stop - start;
    std::cout << "duration: " << duration.count() << std::endl;

    hipMemcpy(h_c, d_c, size*sizeof(int), hipMemcpyDeviceToHost);

#ifdef VERIFY
    verify(n, h_a, h_b, h_c);
#endif

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}