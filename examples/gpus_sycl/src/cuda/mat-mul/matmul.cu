

#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <assert.h>
#include <math.h>
#include <fcntl.h>
#include <unistd.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32 // max 1024 threads per block
#endif

__global__
void matmul(int n, int *a, int *b, int *c) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int tmp = 0;
    for (int i=0; i<n; i++) {
        tmp += a[n*y+i] * b[n*i+x];
    }
    c[y*n+x] = tmp;
}

void verify(int n, int *a, int *b, int *c) {
    for (int y=0; y<n; y++) {
        for (int x=0; x<n; x++) {
            int tmp = 0;
            for (int i=0; i<n; i++) {
                tmp += a[n*y+i] * b[n*i+x];
            }
            assert(tmp == c[y*n+x]);
        }
    }
}

int main(void) {
    int n = 1024;
    int size = n*n;

    // host memory
    int *h_a = (int*)malloc(size*sizeof(int));
    int *h_b = (int*)malloc(size*sizeof(int));
    int *h_c = (int*)calloc(size, sizeof(int));

    // device memory
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size*sizeof(int));
    hipMalloc(&d_b, size*sizeof(int));
    hipMalloc(&d_c, size*sizeof(int));
    float test = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 dimGrid(numBlocks, numBlocks, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    printf("%d %d %f\n", BLOCK_SIZE, numBlocks, test);

    for (int i=0; i<size; i++) {
        h_a[i] = 1;
        h_b[i] = 2;
    }
    // copy over
    hipMemcpy(d_a, h_a, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size*sizeof(int), hipMemcpyHostToDevice);

    // test events
    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);

    auto start = std::chrono::high_resolution_clock::now();
    hipEventRecord(e_start);
    matmul<<<dimGrid, dimBlock>>>(n, d_a, d_b, d_c);
    hipEventRecord(e_stop);
    // cudaDeviceSynchronize();
    hipEventSynchronize(e_stop);

    hipError_t code = hipPeekAtLastError();
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s\n", hipGetErrorString(code));
    }

    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = stop - start;
    std::cout << "duration: " << duration.count()*1000.0 << std::endl;
    // milliseconds
    float kernel_time = 0;
    hipEventElapsedTime(&kernel_time, e_start, e_stop);
    std::cout << "kernel_time: " << kernel_time << std::endl;

    hipMemcpy(h_c, d_c, size*sizeof(int), hipMemcpyDeviceToHost);

#ifdef VERIFY
    verify(n, h_a, h_b, h_c);
#endif

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}